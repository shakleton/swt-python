extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDNN

static int _ccv_nnc_add_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 2);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	const float p = cmd.info.blas.a[0];
	static const float zero = 0;
	if (inputs[1] == 0)
	{
		const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
		const ccv_nnc_cudnn_tensor_view_descriptor_t c = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
		CUDNN_ENFORCE(cudnnTransformTensor(cudnn, &p, a.descriptor, a.data.u8,  &zero, c.descriptor, c.data.u8));
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(c);
		return CCV_NNC_EXEC_SUCCESS;
	}
	const float q = cmd.info.blas.a[1];
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[1]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t c = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	hipdnnOpTensorDescriptor_t add = ccv_nnc_stream_context_get_op_tensor_descriptor(stream_context);
	hipdnnSetOpTensorDescriptor(add, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, add, &p, a.descriptor, a.data.u8, &q, b.descriptor, b.data.u8, &zero, c.descriptor, c.data.u8));
	ccv_nnc_stream_context_return_op_tensor_descriptor(stream_context, add);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(c);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_add_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	const float p = cmd.info.blas.a[0];
	const float q = cmd.info.blas.a[1];
	if (inputs[0] == 0)
	{
		if (outputs[0])
		{
			const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
			CUDNN_ENFORCE(hipdnnSetTensor(cudnn, a.descriptor, a.data.u8, &p));
			ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
		}
		if (output_size > 1 && outputs[1])
		{
			const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[1]);
			CUDNN_ENFORCE(hipdnnSetTensor(cudnn, b.descriptor, b.data.u8, &q));
			ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
		}
		return CCV_NNC_EXEC_SUCCESS;
	}
	int gdim[CCV_NNC_MAX_DIM + 2];
	ccv_nnc_tensor_view_t* const g = (ccv_nnc_tensor_view_t*)inputs[0];
	ccv_nnc_tensor_view_get_dim(g, gdim);
	static const float zero = 0;
	const ccv_nnc_cudnn_tensor_view_descriptor_t gcu = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, g);
	ccv_nnc_tensor_view_t* const a = (ccv_nnc_tensor_view_t*)outputs[0];
	ccv_nnc_cudnn_tensor_view_descriptor_t acu;
	if (a)
		acu = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, a);
	const int reduce_a_dim = a ? !ccv_nnc_tensor_view_check_dim(a, gdim) : 0;
	ccv_nnc_tensor_view_t* const b = output_size > 1 ? (ccv_nnc_tensor_view_t*)outputs[1] : 0;
	ccv_nnc_cudnn_tensor_view_descriptor_t bcu;
	if (b)
		 bcu = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, b);
	const int reduce_b_dim = b ? !ccv_nnc_tensor_view_check_dim(b, gdim) : 0;
	hipdnnReduceTensorDescriptor_t reduce_sum;
	if ((a && reduce_a_dim) || (b && reduce_b_dim))
	{
		reduce_sum = ccv_nnc_stream_context_get_reduce_tensor_descriptor(stream_context);
		hipdnnSetReduceTensorDescriptor(reduce_sum, HIPDNN_REDUCE_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
	}
	size_t workspace_size = 0;
	void* workspace = 0;
	if (a && reduce_a_dim)
	{
		size_t a_workspace_size = 0;
		CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce_sum, gcu.descriptor, acu.descriptor, &a_workspace_size));
		if (a_workspace_size > workspace_size)
			workspace_size = a_workspace_size;
	}
	if (b && reduce_b_dim)
	{
		size_t b_workspace_size = 0;
		CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce_sum, gcu.descriptor, bcu.descriptor, &b_workspace_size));
		if (b_workspace_size > workspace_size)
			workspace_size = b_workspace_size;
	}
	if (workspace_size)
		workspace = ccv_nnc_stream_context_get_workspace(stream_context, workspace_size, CCV_TENSOR_GPU_MEMORY);
	if (a)
	{
		if (reduce_a_dim)
		{
			CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce_sum, 0, 0, workspace, workspace_size, &p, gcu.descriptor, gcu.data.u8, &zero, acu.descriptor, acu.data.u8));
		} else {
			CUDNN_ENFORCE(cudnnTransformTensor(cudnn, &p, gcu.descriptor, gcu.data.u8,  &zero, acu.descriptor, acu.data.u8));
		}
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(acu);
	}
	if (b)
	{
		if (reduce_b_dim)
		{
			CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce_sum, 0, 0, workspace, workspace_size, &q, gcu.descriptor, gcu.data.u8, &zero, bcu.descriptor, bcu.data.u8));
		} else {
			CUDNN_ENFORCE(cudnnTransformTensor(cudnn, &q, gcu.descriptor, gcu.data.u8,  &zero, bcu.descriptor, bcu.data.u8));
		}
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(bcu);
	}
	if ((a && reduce_a_dim) || (b && reduce_b_dim))
		ccv_nnc_stream_context_return_reduce_tensor_descriptor(stream_context, reduce_sum);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(gcu);
	return CCV_NNC_EXEC_SUCCESS;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_ADD_FORWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_add_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_ADD_BACKWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_add_back;
#endif
}

