#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

static inline __device__ __half log(const half v)
{
	return hlog(v);
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_categorical_crossentropy_forw_kernel(const int batch_size, const int count, const NUM1* const label, const NUM2* const a, NUM2* const c)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		const int idx = (int)((float)label[i] + 0.5);
		c[i] = -log(a[i * count + idx]);
	}
}

template<typename NUM>
__global__ void _ccv_nnc_categorical_crossentropy_one_hot_forw_kernel(const int batch_size, const int count, const NUM* const label, const NUM* const a, NUM* const c)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		NUM p = label[i * count] * log(a[i * count]);
		for (int j = 1; j < count; j++)
			p += label[i * count + j] * log(a[i * count + j]);
		c[i] = -p;
	}
}

template<typename NUM>
__global__ void _ccv_nnc_categorical_crossentropy_forw_kernel(const int batch_size, const int count, const int* const label, const NUM* const a, NUM* const c)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		c[i] = -log(a[i * count + label[i]]);
	}
}

static int _ccv_nnc_categorical_crossentropy_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 2);
	const ccv_nnc_tensor_t* a = inputs[0];
	assert(!CCV_IS_TENSOR_VIEW(a));
	const ccv_nnc_tensor_t* b = inputs[1];
	assert(!CCV_IS_TENSOR_VIEW(b));
	assert(output_size == 1);
	ccv_nnc_tensor_t* c = outputs[0];
	assert(!CCV_IS_TENSOR_VIEW(c));
	const int axis_count = ccv_nnc_tensor_nd(a->info.dim);
	const int batch_size = axis_count < 2 ? 1 : a->info.dim[0];
	const int count = ccv_nnc_tensor_count(a->info) / batch_size;
	int i;
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	assert(a->info.datatype == c->info.datatype);
	if (b->info.datatype == CCV_32F || b->info.datatype == CCV_16F)
	{
		// If has more than 1 axis, then the range is the channel count. Otherwise, if our batch size is 1, then the range is
		// the channel count. Otherwise, the range is 1 (and the only axis is the batch size).
		const int range = ccv_nnc_tensor_nd(b->info.dim) > 1 ? ccv_nnc_tensor_get_c(b->info) : (batch_size == 1 ? b->info.dim[0] : 1);
		if (range == 1)
		{
			for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && b->info.dim[i] > 0; i++)
				{ assert(b->info.dim[i] == c->info.dim[i]); }
			if (b->info.datatype == CCV_32F)
			{
				if (a->info.datatype == CCV_16F)
					_ccv_nnc_categorical_crossentropy_forw_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, b->data.f32, (__half*)a->data.f16, (__half*)c->data.f16);
				else
					_ccv_nnc_categorical_crossentropy_forw_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, b->data.f32, a->data.f32, c->data.f32);
			} else {
				assert(b->info.datatype == CCV_16F);
				assert(a->info.datatype == CCV_16F);
				_ccv_nnc_categorical_crossentropy_forw_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, (__half*)b->data.f16, (__half*)a->data.f16, (__half*)c->data.f16);
			}
		} else {
			assert(range == count);
			assert(a->info.datatype == b->info.datatype);
			if (a->info.datatype == CCV_16F)
				_ccv_nnc_categorical_crossentropy_one_hot_forw_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, (__half*)b->data.f16, (__half*)a->data.f16, (__half*)c->data.f16);
			else
				_ccv_nnc_categorical_crossentropy_one_hot_forw_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, b->data.f32, a->data.f32, c->data.f32);
		}
	} else if (b->info.datatype == CCV_32S) {
		for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && b->info.dim[i] > 0; i++)
			{ assert(b->info.dim[i] == c->info.dim[i]); }
		if (a->info.datatype == CCV_16F)
			_ccv_nnc_categorical_crossentropy_forw_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, b->data.i32, (__half*)a->data.f16, (__half*)c->data.f16);
		else
			_ccv_nnc_categorical_crossentropy_forw_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, b->data.i32, a->data.f32, c->data.f32);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

template<typename NUM>
__global__ void _ccv_nnc_set_zero_kernel(const int n, NUM* const a)
{
	CUDA_1D_KERNEL_LOOP(i, n) {
		a[i] = 0;
	}
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_categorical_crossentropy_back_kernel(const int batch_size, const int count, const NUM2* const g, const NUM1* const label, const NUM2* const a, NUM2* const h)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		const int idx = (int)((float)label[i] + 0.5);
		h[i * count + idx] = -g[i] / a[i * count + idx];
	}
}

template<typename NUM>
__global__ void _ccv_nnc_categorical_crossentropy_one_hot_back_kernel(const int batch_size_count, const int count, const NUM* const g, const NUM* const label, const NUM* const a, NUM* const h)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size_count) {
		const int idx = i / count;
		h[i] = -g[idx] * label[i] / a[i];
	}
}

template<typename NUM>
__global__ void _ccv_nnc_categorical_crossentropy_back_kernel(const int batch_size, const int count, const NUM* const g, const int* const label, const NUM* const a, NUM* const h)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		const int idx = label[i];
		h[i * count + idx] = -g[i] / a[i * count + idx];
	}
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_categorical_crossentropy_back_kernel(const int batch_size, const int count, const NUM1* const label, const NUM2* const a, NUM2* const h)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		const int idx = (int)((float)label[i] + 0.5);
		h[i * count + idx] = (NUM2)-1. / a[i * count + idx];
	}
}

template<typename NUM>
__global__ void _ccv_nnc_categorical_crossentropy_one_hot_back_kernel(const int batch_size_count, const int count, const NUM* const label, const NUM* const a, NUM* const h)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size_count) {
		h[i] = -label[i] / a[i];
	}
}

template<typename NUM>
__global__ void _ccv_nnc_categorical_crossentropy_back_kernel(const int batch_size, const int count, const int* const label, const NUM* const a, NUM* const h)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		const int idx = label[i];
		h[i * count + idx] = (NUM)-1. / a[i * count + idx];
	}
}

static int _ccv_nnc_categorical_crossentropy_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size >= 3);
	assert(output_size >= 1);
	const ccv_nnc_tensor_t* g = inputs[0];
	assert(!g || !CCV_IS_TENSOR_VIEW(g));
	const ccv_nnc_tensor_t* a = inputs[1];
	assert(!CCV_IS_TENSOR_VIEW(a));
	const ccv_nnc_tensor_t* b = inputs[2];
	assert(!CCV_IS_TENSOR_VIEW(b));
	ccv_nnc_tensor_t* h = outputs[0];
	assert(!CCV_IS_TENSOR_VIEW(h));
	const int axis_count = ccv_nnc_tensor_nd(a->info.dim);
	const int batch_size = axis_count < 2 ? 1 : a->info.dim[0];
	const int bcount = ccv_nnc_tensor_count(a->info);
	const int count = bcount / batch_size;
	int i;
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	assert(a->info.datatype == h->info.datatype);
	const int datatype = a->info.datatype;
	if (datatype == CCV_16F)
		_ccv_nnc_set_zero_kernel<<<CUDA_GET_BLOCKS(bcount), CUDA_NUM_THREADS, 0, stream>>>(bcount, (__half *)h->data.f16);
	else
		_ccv_nnc_set_zero_kernel<<<CUDA_GET_BLOCKS(bcount), CUDA_NUM_THREADS, 0, stream>>>(bcount, h->data.f32);
	if (g)
	{
		assert(g->info.datatype == datatype);
		if (b->info.datatype == CCV_32F || b->info.datatype == CCV_16F)
		{
			// If has more than 1 axis, then the range is the channel count. Otherwise, if our batch size is 1, then the range is
			// the channel count. Otherwise, the range is 1 (and the only axis is the batch size).
			const int range = ccv_nnc_tensor_nd(b->info.dim) > 1 ? ccv_nnc_tensor_get_c(b->info) : (batch_size == 1 ? b->info.dim[0] : 1);
			if (range == 1)
			{
				for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && a->info.dim[i] > 0; i++)
					{ assert(a->info.dim[i] == h->info.dim[i]); }
				if (b->info.datatype == CCV_32F)
				{
					if (datatype == CCV_16F)
						_ccv_nnc_categorical_crossentropy_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, (__half*)g->data.f16, b->data.f32, (__half*)a->data.f16, (__half*)h->data.f16);
					else
						_ccv_nnc_categorical_crossentropy_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, g->data.f32, b->data.f32, a->data.f32, h->data.f32);
				} else {
					assert(b->info.datatype == CCV_16F);
					assert(datatype == CCV_16F);
					_ccv_nnc_categorical_crossentropy_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, (__half*)g->data.f16, (__half*)b->data.f16, (__half*)a->data.f16, (__half*)h->data.f16);
				}
			} else {
				assert(range == count);
				assert(b->info.datatype == datatype);
				if (datatype == CCV_16F)
					_ccv_nnc_categorical_crossentropy_one_hot_back_kernel<<<CUDA_GET_BLOCKS(bcount), CUDA_NUM_THREADS, 0, stream>>>(bcount, count, (__half*)g->data.f16, (__half*)b->data.f16, (__half*)a->data.f16, (__half*)h->data.f16);
				else
					_ccv_nnc_categorical_crossentropy_one_hot_back_kernel<<<CUDA_GET_BLOCKS(bcount), CUDA_NUM_THREADS, 0, stream>>>(bcount, count, g->data.f32, b->data.f32, a->data.f32, h->data.f32);
			}
		} else if (b->info.datatype == CCV_32S) {
			for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && a->info.dim[i] > 0; i++)
				{ assert(a->info.dim[i] == h->info.dim[i]); }
			if (datatype == CCV_16F)
				_ccv_nnc_categorical_crossentropy_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, (__half*)g->data.f16, b->data.i32, (__half*)a->data.f16, (__half*)h->data.f16);
			else
				_ccv_nnc_categorical_crossentropy_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, g->data.f32, b->data.i32, a->data.f32, h->data.f32);
		}
	} else {
		if (b->info.datatype == CCV_32F || b->info.datatype == CCV_16F)
		{
			// If has more than 1 axis, then the range is the channel count. Otherwise, if our batch size is 1, then the range is
			// the channel count. Otherwise, the range is 1 (and the only axis is the batch size).
			const int range = ccv_nnc_tensor_nd(b->info.dim) > 1 ? ccv_nnc_tensor_get_c(b->info) : (batch_size == 1 ? b->info.dim[0] : 1);
			if (range == 1)
			{
				for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && a->info.dim[i] > 0; i++)
					{ assert(a->info.dim[i] == h->info.dim[i]); }
				if (b->info.datatype == CCV_32F)
				{
					if (datatype == CCV_16F)
						_ccv_nnc_categorical_crossentropy_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, b->data.f32, (__half*)a->data.f16, (__half*)h->data.f16);
					else
						_ccv_nnc_categorical_crossentropy_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, b->data.f32, a->data.f32, h->data.f32);
				} else {
					assert(b->info.datatype == CCV_16F);
					assert(datatype == CCV_16F);
					_ccv_nnc_categorical_crossentropy_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, (__half*)b->data.f16, (__half*)a->data.f16, (__half*)h->data.f16);
				}
			} else {
				assert(range == count);
				assert(b->info.datatype == datatype);
				if (datatype == CCV_16F)
					_ccv_nnc_categorical_crossentropy_one_hot_back_kernel<<<CUDA_GET_BLOCKS(bcount), CUDA_NUM_THREADS, 0, stream>>>(bcount, count, (__half*)b->data.f16, (__half*)a->data.f16, (__half*)h->data.f16);
				else
					_ccv_nnc_categorical_crossentropy_one_hot_back_kernel<<<CUDA_GET_BLOCKS(bcount), CUDA_NUM_THREADS, 0, stream>>>(bcount, count, b->data.f32, a->data.f32, h->data.f32);
			}
		} else if (b->info.datatype == CCV_32S) {
			for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && a->info.dim[i] > 0; i++)
				{ assert(a->info.dim[i] == h->info.dim[i]); }
			if (datatype == CCV_16F)
				_ccv_nnc_categorical_crossentropy_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, b->data.i32, (__half*)a->data.f16, (__half*)h->data.f16);
			else
				_ccv_nnc_categorical_crossentropy_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, b->data.i32, a->data.f32, h->data.f32);
		}
	}
	return CCV_NNC_EXEC_SUCCESS;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_CATEGORICAL_CROSSENTROPY_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_32S | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_categorical_crossentropy_forw;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_CATEGORICAL_CROSSENTROPY_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_32S | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_categorical_crossentropy_back;
}
