extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDNN

static int _ccv_nnc_softmax_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 1);
	assert(output_size == 1);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	static const float one = 1, zero = 0;
	CUDNN_ENFORCE(hipdnnSoftmaxForward(cudnn, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, &one, a.descriptor, a.data.u8, &zero, b.descriptor, b.data.u8));
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_softmax_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 3);
	assert(output_size == 1);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	const ccv_nnc_cudnn_tensor_view_descriptor_t g = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[2]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t h = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	static const float one = 1, zero = 0;
	CUDNN_ENFORCE(hipdnnSoftmaxBackward(cudnn, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, &one, b.descriptor, b.data.u8, g.descriptor, g.data.u8, &zero, h.descriptor, h.data.u8));
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(g);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(h);
	return CCV_NNC_EXEC_SUCCESS;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_SOFTMAX_FORWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_softmax_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_SOFTMAX_BACKWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_softmax_back;
#endif
}
