extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDNN

static int _ccv_nnc_format_transform(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(output_size <= input_size);
	int i;
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	for (i = 0; i < output_size; i++)
	{
		const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[i]);
		const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)outputs[i]);
		static const float one = 1, zero = 0;
		CUDNN_ENFORCE(cudnnTransformTensor(cudnn, &one, a.descriptor, a.data.u8, &zero, b.descriptor, b.data.u8));
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_FORMAT_TRANSFORM_FORWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_format_transform;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_FORMAT_TRANSFORM_BACKWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_format_transform;
#endif
}

#ifdef HAVE_CUDNN

static int _ccv_nnc_set_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 0);
	int i;
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	for (i = 0; i < output_size; i++)
	{
		const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[i]);
		CUDNN_ENFORCE(hipdnnSetTensor(cudnn, a.descriptor, a.data.u8, &cmd.info.blas.a[0]));
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_set_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	int i;
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	for (i = 0; i < output_size; i++)
	{
		const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[i]);
		static const float zero = 0;
		CUDNN_ENFORCE(hipdnnSetTensor(cudnn, a.descriptor, a.data.u8, &zero));
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_SET_FORWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_set_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_SET_BACKWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_set_back;
#endif
}
