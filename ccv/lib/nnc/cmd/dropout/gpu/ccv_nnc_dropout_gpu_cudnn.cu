extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDNN

static int _ccv_nnc_dropout_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 1);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	assert(output_size == 2);
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	hipdnnDropoutDescriptor_t dropout = ccv_nnc_stream_context_get_dropout_descriptor(stream_context, cmd.info.dropout.p);
	ccv_nnc_tensor_t* const mask = outputs[1];
	assert(!CCV_IS_TENSOR_VIEW(mask));
	const int tensor_count = ccv_nnc_tensor_count(mask->info);
	const size_t reserved_size = CCV_GET_DATA_TYPE_SIZE(mask->info.datatype) * tensor_count;
	CUDNN_ENFORCE(cudnnDropoutForward(cudnn, dropout, a.descriptor, a.data.u8, b.descriptor, b.data.u8, mask->data.u8, reserved_size));
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
	ccv_nnc_stream_context_return_dropout_descriptor(stream_context, dropout);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_dropout_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 5);
	assert(output_size >= 1);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	const ccv_nnc_cudnn_tensor_view_descriptor_t g = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t h = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	hipdnnDropoutDescriptor_t dropout = ccv_nnc_stream_context_get_dropout_descriptor(stream_context, cmd.info.dropout.p);
	ccv_nnc_tensor_t* const mask = inputs[4];
	assert(!CCV_IS_TENSOR_VIEW(mask));
	const int tensor_count = ccv_nnc_tensor_count(mask->info);
	const size_t reserved_size = CCV_GET_DATA_TYPE_SIZE(mask->info.datatype) * tensor_count;
	CUDNN_ENFORCE(cudnnDropoutBackward(cudnn, dropout, g.descriptor, g.data.u8, h.descriptor, h.data.u8, mask->data.u8, reserved_size));
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(g);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(h);
	ccv_nnc_stream_context_return_dropout_descriptor(stream_context, dropout);
	return CCV_NNC_EXEC_SUCCESS;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_DROPOUT_FORWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_dropout_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_DROPOUT_BACKWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_dropout_back;
#endif
}
