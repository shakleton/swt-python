#include "hip/hip_runtime.h"
#include "ccv_nnc_compat.h"
extern "C" {
#include <nnc/ccv_nnc_easy.h>
#include <nnc/_ccv_nnc_stream.h>
}

void* cumalloc(int device, size_t size)
{
	void* ptr = 0;
	hipSetDevice(device);
	hipMalloc(&ptr, size);
	return ptr;
}

void cufree(int device, void* ptr)
{
	hipSetDevice(device);
	hipFree(ptr);
}

void cudevice(int device)
{
	if (device >= 0)
		hipSetDevice(device);
}

void cumemcpy(void* dest, const int dest_type, const void* src, const int src_type, size_t n)
{
	if (CCV_TENSOR_GET_MEMORY(src_type) == CCV_TENSOR_CPU_MEMORY && CCV_TENSOR_GET_MEMORY(dest_type) == CCV_TENSOR_GPU_MEMORY) {
		const int device_b = CCV_TENSOR_GET_DEVICE_ID(dest_type);
		hipSetDevice(device_b);
		CUDA_ENFORCE(hipMemcpy(dest, src, n, hipMemcpyHostToDevice));
	} else if (CCV_TENSOR_GET_MEMORY(src_type) == CCV_TENSOR_GPU_MEMORY && CCV_TENSOR_GET_MEMORY(dest_type) == CCV_TENSOR_CPU_MEMORY) {
		const int device_a = CCV_TENSOR_GET_DEVICE_ID(src_type);
		hipSetDevice(device_a);
		CUDA_ENFORCE(hipMemcpy(dest, src, n, hipMemcpyDeviceToHost));
	} else if (CCV_TENSOR_GET_MEMORY(src_type) == CCV_TENSOR_CPU_MEMORY && CCV_TENSOR_GET_MEMORY(dest_type) == CCV_TENSOR_CPU_MEMORY)
		CUDA_ENFORCE(hipMemcpy(dest, src, n, hipMemcpyHostToHost));
	else if (CCV_TENSOR_GET_MEMORY(src_type) == CCV_TENSOR_GPU_MEMORY && CCV_TENSOR_GET_MEMORY(dest_type) == CCV_TENSOR_GPU_MEMORY) {
		const int device_a = CCV_TENSOR_GET_DEVICE_ID(src_type);
		const int device_b = CCV_TENSOR_GET_DEVICE_ID(dest_type);
		hipSetDevice(device_b);
		if (device_a == device_b)
			CUDA_ENFORCE(hipMemcpy(dest, src, n, hipMemcpyDeviceToDevice));
		else
			CUDA_ENFORCE(hipMemcpyPeer(dest, device_b, src, device_a, n));
	}
}

void* cuhostalloc(size_t size)
{
	void* ptr = 0;
	hipHostAlloc(&ptr, size, hipHostMallocPortable);
	return ptr;
}

void cuhostfree(void* ptr)
{
	hipHostFree(ptr);
}

int curegister(void* ptr, size_t size)
{
	return hipSuccess == hipHostRegister(ptr, size, hipHostRegisterPortable);
}

void cuunregister(void* ptr)
{
	hipHostUnregister(ptr);
}

typedef struct {
	hipStream_t stream;
	hipblasHandle_t cublas;
	struct {
		int n;
		__half* data;
	} ones_16;
	struct {
		int n;
		float* data;
	} ones_32;
	struct {
		int n;
		double* data;
	} ones_64;
	size_t workspace_size;
	void* workspace;
#ifdef HAVE_CUDNN
	hipdnnHandle_t cudnn;
	void* rngs; // user-allocated GPU memory that will hold random number generator states.
#endif
} ccv_nnc_stream_context_device_local_t;

typedef struct {
#ifdef HAVE_NCCL
	ncclComm_t* comms;
	int comm_count;
#endif
} ccv_nnc_stream_resource_container_compat_t;

typedef struct {
	ccv_nnc_stream_context_t super;
	struct {
		size_t workspace_size;
		void* workspace;
	} cpu;
	unsigned long long seed;
	union {
		ccv_nnc_stream_context_device_local_t _inline_gpu;
		struct {
			ccv_nnc_stream_context_device_local_t* _heap_gpus;
			int _heap_gpu_size;
		};
	};
} ccv_nnc_stream_context_compat_t;

static ccv_nnc_stream_context_device_local_t* _ccv_nnc_stream_compat_device_local(ccv_nnc_stream_context_compat_t* const stream_compat)
{
	int device_id = CCV_STREAM_GET_DEVICE_ID(stream_compat->super.type);
	if (device_id == CCV_STREAM_GET_DEVICE_ID(CCV_COMPUTE_DEVICE_ANY))
	{
		hipGetDevice(&device_id);
		if (stream_compat->_heap_gpu_size <= device_id)
		{
			if (!stream_compat->_heap_gpus)
				stream_compat->_heap_gpus = (ccv_nnc_stream_context_device_local_t*)cccalloc(device_id + 1, sizeof(ccv_nnc_stream_context_device_local_t));
			else {
				stream_compat->_heap_gpus = (ccv_nnc_stream_context_device_local_t*)ccrealloc(stream_compat->_heap_gpus, sizeof(ccv_nnc_stream_context_device_local_t) * (device_id + 1));
				memset(stream_compat->_heap_gpus + stream_compat->_heap_gpu_size, 0, sizeof(ccv_nnc_stream_context_device_local_t) * (device_id + 1 - stream_compat->_heap_gpu_size));
			}
			stream_compat->_heap_gpu_size = device_id + 1;
		}
		return stream_compat->_heap_gpus + device_id;
	} else {
		hipSetDevice(device_id);
		return &stream_compat->_inline_gpu;
	}
}

static ccv_nnc_stream_context_compat_t* _ccv_nnc_default_stream_compat()
{
	static __thread ccv_nnc_stream_context_compat_t ccv_nnc_per_thread_gpu_stream_context = {
		.super = {
			.type = CCV_STREAM_CONTEXT_GPU | CCV_COMPUTE_DEVICE_ANY,
		},
	};
	return &ccv_nnc_per_thread_gpu_stream_context;
}

typedef struct {
	ccv_nnc_stream_signal_t super;
	hipEvent_t event;
} ccv_nnc_stream_compat_signal_t;

ccv_nnc_stream_signal_t* ccv_nnc_init_stream_signal(ccv_nnc_stream_signal_t* const signal)
{
	assert(CCV_STREAM_GET_CONTEXT(((int*)signal)[0]) == CCV_STREAM_CONTEXT_GPU);
	ccv_nnc_stream_compat_signal_t* compat_signal = (ccv_nnc_stream_compat_signal_t*)ccrealloc(signal, sizeof(ccv_nnc_stream_compat_signal_t));
	const int device = CCV_STREAM_GET_DEVICE_ID(compat_signal->super.type);
	hipSetDevice(device);
	hipEventCreateWithFlags(&compat_signal->event, hipEventDisableTiming);
	return (ccv_nnc_stream_signal_t*)compat_signal;
}

void ccv_nnc_stream_compat_emit_signal(const ccv_nnc_stream_context_t* const stream, const ccv_nnc_stream_signal_t* const signal)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream;
	if (!stream_compat)
		stream_compat = _ccv_nnc_default_stream_compat();
	ccv_nnc_stream_context_device_local_t* const device_local = _ccv_nnc_stream_compat_device_local(stream_compat);
	ccv_nnc_stream_compat_signal_t* compat_signal = (ccv_nnc_stream_compat_signal_t*)signal;
	hipEventRecord(compat_signal->event, device_local->stream);
}

void ccv_nnc_stream_compat_wait_signal(const ccv_nnc_stream_context_t* const stream, const ccv_nnc_stream_signal_t* const signal)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream;
	if (!stream_compat)
		stream_compat = _ccv_nnc_default_stream_compat();
	ccv_nnc_stream_context_device_local_t* const device_local = _ccv_nnc_stream_compat_device_local(stream_compat);
	ccv_nnc_stream_compat_signal_t* compat_signal = (ccv_nnc_stream_compat_signal_t*)signal;
	hipStreamWaitEvent(device_local->stream, compat_signal->event, 0);
}

void ccv_nnc_deinit_stream_signal(ccv_nnc_stream_signal_t* const signal)
{
	ccv_nnc_stream_compat_signal_t* compat_signal = (ccv_nnc_stream_compat_signal_t*)signal;
	const int device = CCV_STREAM_GET_DEVICE_ID(compat_signal->super.type);
	hipSetDevice(device);
	hipEventDestroy(compat_signal->event);
}

int ccv_nnc_gpu_device_count(void)
{
	int count = 0;
	hipGetDeviceCount(&count);
	return count;
}

ccv_nnc_stream_context_t* ccv_nnc_init_stream_context(ccv_nnc_stream_context_t* const stream_context)
{
	assert(CCV_STREAM_GET_CONTEXT(((int*)stream_context)[0]) == CCV_STREAM_CONTEXT_GPU);
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)ccrealloc(stream_context, sizeof(ccv_nnc_stream_context_compat_t));
	const ccv_nnc_stream_context_t super = stream_compat->super;
	memset(stream_compat, 0, sizeof(ccv_nnc_stream_context_compat_t));
	stream_compat->super = super;
	ccv_nnc_stream_context_device_local_t* const device_local = _ccv_nnc_stream_compat_device_local(stream_compat);
	hipStreamCreate(&device_local->stream);
	return (ccv_nnc_stream_context_t*)stream_compat;
}

void* ccv_nnc_stream_compat_get_workspace(const ccv_nnc_stream_context_t* const stream_context, const size_t workspace_size, const int mem)
{
	if (workspace_size <= 0)
		return 0;
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat)
		stream_compat = _ccv_nnc_default_stream_compat();
	if (mem == CCV_TENSOR_CPU_MEMORY)
	{
		if (stream_compat->cpu.workspace_size >= workspace_size)
			return stream_compat->cpu.workspace;
		stream_compat->cpu.workspace_size = workspace_size;
		if (stream_compat->cpu.workspace)
			ccfree(stream_compat->cpu.workspace);
		stream_compat->cpu.workspace = 0;
		const int success = ccmemalign(&stream_compat->cpu.workspace, 16, workspace_size);
		return success != 0 ? 0 : stream_compat->cpu.workspace;
	} else if (mem == CCV_TENSOR_GPU_MEMORY) {
		ccv_nnc_stream_context_device_local_t* const device_local = _ccv_nnc_stream_compat_device_local(stream_compat);
		if (device_local->workspace_size >= workspace_size)
			return device_local->workspace;
		device_local->workspace_size = workspace_size;
		if (device_local->workspace)
			hipFree(device_local->workspace);
		device_local->workspace = 0;
		hipMalloc(&device_local->workspace, workspace_size);
		return device_local->workspace;
	}
	return 0;
}

void ccv_nnc_stream_compat_drain(ccv_nnc_stream_context_t* const stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat)
		stream_compat = _ccv_nnc_default_stream_compat();
	if (stream_compat->cpu.workspace)
	{
		ccfree(stream_compat->cpu.workspace);
		stream_compat->cpu.workspace = 0;
		stream_compat->cpu.workspace_size = 0;
	}
	const int device = CCV_STREAM_GET_DEVICE_ID(stream_compat->super.type);
	if (device == CCV_STREAM_GET_DEVICE_ID(CCV_COMPUTE_DEVICE_ANY))
	{
		int i;
		for (i = 0; i < stream_compat->_heap_gpu_size; i++)
			if (stream_compat->_heap_gpus[i].workspace)
			{
				hipSetDevice(i);
				hipFree(stream_compat->_heap_gpus[i].workspace);
				stream_compat->_heap_gpus[i].workspace = 0;
				stream_compat->_heap_gpus[i].workspace_size = 0;
			}
	} else if (stream_compat->_inline_gpu.workspace) {
		hipSetDevice(device);
		hipFree(stream_compat->_inline_gpu.workspace);
		stream_compat->_inline_gpu.workspace = 0;
		stream_compat->_inline_gpu.workspace_size = 0;
	}
}

void ccv_nnc_synchronize_stream_context(const ccv_nnc_stream_context_t* const stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat)
		stream_compat = _ccv_nnc_default_stream_compat();
	ccv_nnc_stream_context_device_local_t* const device_local = _ccv_nnc_stream_compat_device_local(stream_compat);
	hipStreamSynchronize(device_local->stream);
}

#if CUDA_VERSION >= 10000
static void _ccv_nnc_stream_compat_task_resume(void* userdata)
#else
static void _ccv_nnc_stream_compat_task_resume(hipStream_t stream, hipError_t status, void* userdata)
#endif
{
	ccv_nnc_stream_task_t* const task = (ccv_nnc_stream_task_t*)userdata;
	ccv_nnc_stream_scheduler_t* const scheduler = task->super;
	pthread_mutex_lock(&scheduler->mutex);
	ccv_nnc_stream_scheduler_prepend_task(scheduler, task);
	--scheduler->stream_wait_task_count;
	pthread_cond_signal(&scheduler->wait);
	pthread_mutex_unlock(&scheduler->mutex);
}

void ccv_nnc_stream_compat_task_synchronize(ccv_nnc_stream_task_t* const self, ccv_nnc_stream_context_t* const stream)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream;
	ccv_nnc_stream_context_device_local_t* const device_local = _ccv_nnc_stream_compat_device_local(stream_compat);
	// If the stream is completed, no need to wait.
	if (hipStreamQuery(device_local->stream) == hipSuccess)
		return;
	ccv_nnc_stream_scheduler_t* const scheduler = self->super;
	pthread_mutex_lock(&scheduler->mutex);
	++scheduler->stream_wait_task_count;
#if CUDA_VERSION >= 10000
	hipLaunchHostFunc(device_local->stream, _ccv_nnc_stream_compat_task_resume, self);
#else
	hipStreamAddCallback(device_local->stream, _ccv_nnc_stream_compat_task_resume, self, 0);
#endif
	pthread_mutex_unlock(&scheduler->mutex);
	swapcontext(&scheduler->callee, &scheduler->caller);
}

void ccv_nnc_deinit_stream_context(ccv_nnc_stream_context_t* const stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (stream_compat->cpu.workspace)
		ccfree(stream_compat->cpu.workspace);
	const int device = CCV_STREAM_GET_DEVICE_ID(stream_compat->super.type);
	if (device == CCV_STREAM_GET_DEVICE_ID(CCV_COMPUTE_DEVICE_ANY))
	{
		int i;
		for (i = 0; i < stream_compat->_heap_gpu_size; i++)
			if (stream_compat->_heap_gpus[i].workspace)
			{
				hipSetDevice(i);
				if (stream_compat->_heap_gpus[i].workspace)
					hipFree(stream_compat->_heap_gpus[i].workspace);
				hipStreamDestroy(stream_compat->_heap_gpus[i].stream);
				if (stream_compat->_heap_gpus[i].cublas)
					hipblasDestroy(stream_compat->_heap_gpus[i].cublas);
				if (stream_compat->_heap_gpus[i].ones_16.data)
					hipFree(stream_compat->_heap_gpus[i].ones_16.data);
				if (stream_compat->_heap_gpus[i].ones_32.data)
					hipFree(stream_compat->_heap_gpus[i].ones_32.data);
				if (stream_compat->_heap_gpus[i].ones_64.data)
					hipFree(stream_compat->_heap_gpus[i].ones_64.data);
#ifdef HAVE_CUDNN
				if (stream_compat->_heap_gpus[i].cudnn)
					hipdnnDestroy(stream_compat->_heap_gpus[i].cudnn);
				if (stream_compat->_heap_gpus[i].rngs)
					hipFree(stream_compat->_heap_gpus[i].rngs);
#endif
			}
	} else {
		hipSetDevice(device);
		if (stream_compat->_inline_gpu.workspace)
			hipFree(stream_compat->_inline_gpu.workspace);
		hipStreamDestroy(stream_compat->_inline_gpu.stream);
		if (stream_compat->_inline_gpu.cublas)
			hipblasDestroy(stream_compat->_inline_gpu.cublas);
		if (stream_compat->_inline_gpu.ones_16.data)
			hipFree(stream_compat->_inline_gpu.ones_16.data);
		if (stream_compat->_inline_gpu.ones_32.data)
			hipFree(stream_compat->_inline_gpu.ones_32.data);
		if (stream_compat->_inline_gpu.ones_64.data)
			hipFree(stream_compat->_inline_gpu.ones_64.data);
#ifdef HAVE_CUDNN
		if (stream_compat->_inline_gpu.cudnn)
			hipdnnDestroy(stream_compat->_inline_gpu.cudnn);
		if (stream_compat->_inline_gpu.rngs)
			hipFree(stream_compat->_inline_gpu.rngs);
#endif
	}
#ifdef HAVE_NCCL
	if (stream_compat->super._inline_container[0])
	{
		ccv_nnc_stream_resource_container_compat_t* const resource_container_compat = (ccv_nnc_stream_resource_container_compat_t*)stream_compat->super._inline_container[0];
		if (resource_container_compat->comms)
		{
			int i;
			for (i = 0; i < resource_container_compat->comm_count; i++)
				NCCL_ENFORCE(ncclCommDestroy(resource_container_compat->comms[i]));
		}
		ccfree(resource_container_compat);
	}
#endif
}

int ccv_nnc_stream_context_get_device(const ccv_nnc_stream_context_t* const stream_context)
{
	if (!stream_context)
	{
		int device = 0;
		hipGetDevice(&device);
		return device;
	}
	const ccv_nnc_stream_context_compat_t* stream_compat = (const ccv_nnc_stream_context_compat_t*)stream_context;
	return CCV_STREAM_GET_DEVICE_ID(stream_compat->super.type);
}

hipStream_t ccv_nnc_stream_context_get_stream(const ccv_nnc_stream_context_t* const stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat)
		stream_compat = _ccv_nnc_default_stream_compat();
	ccv_nnc_stream_context_device_local_t* const device_local = _ccv_nnc_stream_compat_device_local(stream_compat);
	return device_local->stream;
}

hipblasHandle_t ccv_nnc_stream_context_get_cublas(const ccv_nnc_stream_context_t* const stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat)
		stream_compat = _ccv_nnc_default_stream_compat();
	ccv_nnc_stream_context_device_local_t* const device_local = _ccv_nnc_stream_compat_device_local(stream_compat);
	if (!device_local->cublas)
	{
		hipblasCreate(&device_local->cublas);
		hipblasSetStream(device_local->cublas, device_local->stream);
	}
	return device_local->cublas;
}

// A simple kernel to set all values to 1.
template<typename NUM>
__global__ static void _ones(NUM* x, int n)
{
	const int thidx = blockIdx.x * blockDim.x + threadIdx.x;
	if (thidx < n)
		x[thidx] = 1.;
}

template<typename ONES>
static void* _ccv_nnc_stream_context_get_ones(ONES &device_ones, const int n, hipStream_t &stream)
{
	if (!device_ones.data || n > device_ones.n)
	{
		if (device_ones.data)
			hipFree(device_ones.data);
		device_ones.n = n;
		CUDA_ENFORCE(hipMalloc(&device_ones.data, sizeof(device_ones.data[0]) * n));
		const int block_x = (n + 255) >> 8;
		_ones<<<block_x, 256, 0, stream>>>(device_ones.data, n);
	}
	return device_ones.data;
}

void* ccv_nnc_stream_context_get_ones(const ccv_nnc_stream_context_t* const stream_context, const int n, const int datatype)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat)
		stream_compat = _ccv_nnc_default_stream_compat();
	ccv_nnc_stream_context_device_local_t* const device_local = _ccv_nnc_stream_compat_device_local(stream_compat);
	switch (datatype)
	{
		case CCV_16F:
			return _ccv_nnc_stream_context_get_ones(device_local->ones_16, n, device_local->stream);
		case CCV_64F:
			return _ccv_nnc_stream_context_get_ones(device_local->ones_64, n, device_local->stream);
		case CCV_32F:
		default:
			return _ccv_nnc_stream_context_get_ones(device_local->ones_32, n, device_local->stream);
	}
}

hipDataType ccv_nnc_cuda_datatype(const int datatype)
{
	switch (datatype)
	{
		case CCV_8U:
			return HIP_R_8I;
		case CCV_32S:
			return HIP_R_32F;
		case CCV_16F:
			return HIP_R_16F;
		case CCV_32F:
			return HIP_R_32F;
		case CCV_64F:
			return HIP_R_64F;
	}
	return HIP_R_32F;
}

hipDataType ccv_nnc_cuda_compute_datatype(const int datatype)
{
	switch (datatype)
	{
		case CCV_8U:
		case CCV_32S:
		case CCV_16F:
		case CCV_32F:
			return HIP_R_32F;
		case CCV_64F:
			return HIP_R_64F;
	}
	return HIP_R_32F;
}

#ifdef HAVE_CUDNN

hipdnnDataType_t ccv_nnc_cudnn_datatype(const int datatype)
{
	switch (datatype)
	{
		case CCV_8U:
			return HIPDNN_DATA_INT8;
		case CCV_32S:
			return HIPDNN_DATA_INT32;
		case CCV_16F:
			return HIPDNN_DATA_HALF;
		case CCV_32F:
			return HIPDNN_DATA_FLOAT;
		case CCV_64F:
			return HIPDNN_DATA_DOUBLE;
	}
	return HIPDNN_DATA_FLOAT;
}

hipdnnHandle_t ccv_nnc_stream_context_get_cudnn(const ccv_nnc_stream_context_t* const stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat)
		stream_compat = _ccv_nnc_default_stream_compat();
	ccv_nnc_stream_context_device_local_t* const device_local = _ccv_nnc_stream_compat_device_local(stream_compat);
	if (!device_local->cudnn)
	{
		CUDNN_ENFORCE(hipdnnCreate(&device_local->cudnn));
		CUDNN_ENFORCE(hipdnnSetStream(device_local->cudnn, device_local->stream));
	}
	return device_local->cudnn;
}

hipdnnActivationDescriptor_t ccv_nnc_stream_context_get_activation_descriptor(const ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnActivationDescriptor_t desc;
	hipdnnCreateActivationDescriptor(&desc);
	return desc;
}

hipdnnConvolutionDescriptor_t ccv_nnc_stream_context_get_convolution_descriptor(const ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnConvolutionDescriptor_t desc;
	hipdnnCreateConvolutionDescriptor(&desc);
	return desc;
}

hipdnnDropoutDescriptor_t ccv_nnc_stream_context_get_dropout_descriptor(const ccv_nnc_stream_context_t* const stream_context, const float p)
{
	hipdnnDropoutDescriptor_t desc;
	hipdnnCreateDropoutDescriptor(&desc);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat)
		stream_compat = _ccv_nnc_default_stream_compat();
	ccv_nnc_stream_context_device_local_t* const device_local = _ccv_nnc_stream_compat_device_local(stream_compat);
	size_t state_size;
	hipdnnDropoutGetStatesSize(cudnn, &state_size);
	if (device_local->rngs)
	{
#if CUDNN_VERSION >= 7100
		cudnnRestoreDropoutDescriptor(desc, cudnn, p, device_local->rngs, state_size, stream_compat->seed);
#else
		++stream_compat->seed;
		hipdnnSetDropoutDescriptor(desc, cudnn, p, device_local->rngs, state_size, stream_compat->seed);
#endif
	} else {
		CUDA_ENFORCE(hipMalloc(&device_local->rngs, state_size));
		stream_compat->seed = (unsigned long long)stream_compat;
		hipdnnSetDropoutDescriptor(desc, cudnn, p, device_local->rngs, state_size, stream_compat->seed);
	}
	return desc;
}

hipdnnFilterDescriptor_t ccv_nnc_stream_context_get_filter_descriptor(const ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnFilterDescriptor_t desc;
	hipdnnCreateFilterDescriptor(&desc);
	return desc;
}

hipdnnOpTensorDescriptor_t ccv_nnc_stream_context_get_op_tensor_descriptor(const ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnOpTensorDescriptor_t desc;
	hipdnnCreateOpTensorDescriptor(&desc);
	return desc;
}

hipdnnPoolingDescriptor_t ccv_nnc_stream_context_get_pooling_descriptor(const ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnPoolingDescriptor_t desc;
	hipdnnCreatePoolingDescriptor(&desc);
	return desc;
}

hipdnnReduceTensorDescriptor_t ccv_nnc_stream_context_get_reduce_tensor_descriptor(const ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnReduceTensorDescriptor_t desc;
	hipdnnCreateReduceTensorDescriptor(&desc);
	return desc;
}

hipdnnTensorDescriptor_t ccv_nnc_stream_context_get_tensor_descriptor(const ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnTensorDescriptor_t desc;
	hipdnnCreateTensorDescriptor(&desc);
	return desc;
}

void ccv_nnc_stream_context_return_activation_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnActivationDescriptor_t activation_desc)
{
	hipdnnDestroyActivationDescriptor(activation_desc);
}

void ccv_nnc_stream_context_return_convolution_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnConvolutionDescriptor_t convolution_desc)
{
	hipdnnDestroyConvolutionDescriptor(convolution_desc);
}

void ccv_nnc_stream_context_return_dropout_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnDropoutDescriptor_t dropout_desc)
{
	hipdnnDestroyDropoutDescriptor(dropout_desc);
}

void ccv_nnc_stream_context_return_filter_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnFilterDescriptor_t filter_desc)
{
	hipdnnDestroyFilterDescriptor(filter_desc);
}

void ccv_nnc_stream_context_return_op_tensor_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnOpTensorDescriptor_t op_tensor_desc)
{
	hipdnnDestroyOpTensorDescriptor(op_tensor_desc);
}

void ccv_nnc_stream_context_return_pooling_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnPoolingDescriptor_t pooling_desc)
{
	hipdnnDestroyPoolingDescriptor(pooling_desc);
}

void ccv_nnc_stream_context_return_reduce_tensor_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnReduceTensorDescriptor_t reduce_tensor_desc)
{
	hipdnnDestroyReduceTensorDescriptor(reduce_tensor_desc);
}

void ccv_nnc_stream_context_return_tensor_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnTensorDescriptor_t tensor_desc)
{
	hipdnnDestroyTensorDescriptor(tensor_desc);
}

ccv_nnc_cudnn_tensor_view_descriptor_t ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(const ccv_nnc_stream_context_t* const stream_context, const ccv_nnc_tensor_view_t* const tensor)
{
	ccv_nnc_cudnn_tensor_view_descriptor_t tensor_desc = {
		stream_context,
		ccv_nnc_stream_context_get_tensor_descriptor(stream_context),
		tensor->data,
	};
	int dim[CCV_NNC_MAX_DIM_ALLOC] = {};
	int stride[CCV_NNC_MAX_DIM_ALLOC] = {};
	const int axis_count = ccv_nnc_tensor_nd(tensor->info.dim);
	const int* const inc = CCV_IS_TENSOR_VIEW(tensor) ? tensor->inc : tensor->info.dim;
	int i;
	for (i = axis_count; i < CCV_NNC_MAX_DIM + 2; i++)
		dim[i] = stride[i] = 1;
	dim[axis_count - 1] = tensor->info.dim[axis_count - 1];
	stride[axis_count - 1] = 1;
	for (i = axis_count - 2; i >= 0; i--)
	{
		dim[i] = tensor->info.dim[i];
		stride[i] = stride[i + 1] * inc[i + 1];
	}
	if (axis_count <= 4)
	{
		CUDNN_ENFORCE(hipdnnSetTensor4dDescriptorEx(tensor_desc.descriptor, ccv_nnc_cudnn_datatype(tensor->info.datatype), dim[0], dim[1], dim[2], dim[3], stride[0], stride[1], stride[2], stride[3]));
	} else {
		CUDNN_ENFORCE(hipdnnSetTensorNdDescriptor(tensor_desc.descriptor, ccv_nnc_cudnn_datatype(tensor->info.datatype), axis_count, dim, stride));
	}
	return tensor_desc;
}

ccv_nnc_cudnn_tensor_view_descriptor_t ccv_nnc_cudnn_get_tensor_view_descriptor(const ccv_nnc_stream_context_t* const stream_context, const ccv_nnc_tensor_view_t* const tensor)
{
	ccv_nnc_cudnn_tensor_view_descriptor_t tensor_desc = {
		stream_context,
		ccv_nnc_stream_context_get_tensor_descriptor(stream_context),
		tensor->data,
	};
	// Fill up dimensions with 1s.
	int dim[CCV_NNC_MAX_DIM_ALLOC] = {};
	int stride[CCV_NNC_MAX_DIM_ALLOC] = {};
	const int axis_count = ccv_nnc_tensor_nd(tensor->info.dim);
	const int* const inc = CCV_IS_TENSOR_VIEW(tensor) ? tensor->inc : tensor->info.dim;
	int i;
	if (tensor->info.format == CCV_TENSOR_FORMAT_NCHW)
	{
		switch (axis_count)
		{
			case 1:
				dim[0] = dim[2] = dim[3] = 1;
				dim[1] = tensor->info.dim[0];
				stride[0] = inc[0];
				stride[1] = 1;
				for (i = 2; i < CCV_NNC_MAX_DIM + 2; i++)
					stride[i] = 1;
				break;
			case 2:
				dim[0] = tensor->info.dim[0];
				dim[1] = tensor->info.dim[1];
				dim[2] = dim[3] = 1;
				stride[0] = inc[1];
				stride[1] = 1;
				for (i = 2; i < CCV_NNC_MAX_DIM + 2; i++)
					stride[i] = 1;
				break;
			case CCV_NNC_MAX_DIM + 1:
				dim[0] = 1;
				dim[1] = tensor->info.dim[0];
				stride[CCV_NNC_MAX_DIM + 1] = 1;
				for (i = CCV_NNC_MAX_DIM - 1; i >= 0; i--)
				{
					dim[i + 2] = tensor->info.dim[i + 1];
					stride[i + 1] = stride[i + 2] * inc[i + 1];
				}
				stride[0] = stride[1] * inc[0];
				break;
			case CCV_NNC_MAX_DIM + 2:
				stride[CCV_NNC_MAX_DIM + 1] = 1;
				dim[CCV_NNC_MAX_DIM + 1] = tensor->info.dim[CCV_NNC_MAX_DIM + 1];
				for (i = CCV_NNC_MAX_DIM; i >= 0; i--)
				{
					dim[i] = tensor->info.dim[i];
					stride[i] = stride[i + 1] * inc[i + 1];
				}
				break;
			default:
				assert(0);
		}
	} else if (tensor->info.format == CCV_TENSOR_FORMAT_NHWC) {
		switch (axis_count)
		{
			case 1:
				dim[0] = dim[2] = dim[3] = 1;
				dim[1] = tensor->info.dim[0];
				stride[0] = inc[0];
				stride[1] = 1;
				for (i = 2; i < CCV_NNC_MAX_DIM + 2; i++)
					stride[i] = 1; // Even though technically this should be inc[1] (because hw is after c), however, make it 1 doesn't have any differences and more versatile.
				break;
			case 2:
				dim[0] = tensor->info.dim[0];
				dim[1] = tensor->info.dim[1];
				dim[2] = dim[3] = 1;
				stride[0] = inc[1];
				stride[1] = 1;
				for (i = 2; i < CCV_NNC_MAX_DIM + 2; i++)
					stride[i] = 1; // Even though technically this should be inc[1] (because hw is after c), however, make it 1 doesn't have any differences and more versatile.
				break;
			case CCV_NNC_MAX_DIM + 1:
				dim[0] = 1;
				dim[1] = tensor->info.dim[CCV_NNC_MAX_DIM];
				stride[1] = 1;
				for (i = CCV_NNC_MAX_DIM - 1; i >= 0; i--)
				{
					dim[i + 2] = tensor->info.dim[i];
					stride[i + 2] = (i == CCV_NNC_MAX_DIM - 1) ? inc[i + 1] : stride[i + 3] * inc[i + 1];
				}
				stride[0] = stride[2] * inc[0];
				break;
			case CCV_NNC_MAX_DIM + 2:
				dim[0] = tensor->info.dim[0];
				dim[1] = tensor->info.dim[CCV_NNC_MAX_DIM + 1];
				stride[1] = 1;
				for (i = CCV_NNC_MAX_DIM - 1; i >= 0; i--)
				{
					dim[i + 2] = tensor->info.dim[i + 1];
					stride[i + 2] = (i == CCV_NNC_MAX_DIM - 1) ? inc[i + 2] : stride[i + 3] * inc[i + 2];
				}
				stride[0] = stride[2] * inc[1];
				break;
			default:
				assert(0);
		}
	} else if (tensor->info.format == CCV_TENSOR_FORMAT_CHWN) {
		switch (axis_count)
		{
			case 1:
				dim[0] = dim[2] = dim[3] = 1;
				dim[1] = tensor->info.dim[0];
				stride[0] = inc[0];
				stride[1] = 1;
				for (i = 2; i < CCV_NNC_MAX_DIM + 2; i++)
					stride[i] = 1;
				break;
			case 2:
				dim[0] = tensor->info.dim[1];
				dim[1] = tensor->info.dim[0];
				dim[2] = dim[3] = 1;
				stride[0] = 1;
				stride[1] = inc[1];
				for (i = 2; i < CCV_NNC_MAX_DIM + 2; i++)
					stride[i] = inc[1];
				break;
			case CCV_NNC_MAX_DIM + 1:
				dim[0] = 1;
				dim[1] = tensor->info.dim[0];
				stride[CCV_NNC_MAX_DIM + 1] = 1;
				for (i = CCV_NNC_MAX_DIM - 1; i >= 0; i--)
				{
					dim[i + 2] = tensor->info.dim[i + 1];
					stride[i + 1] = stride[i + 2] * inc[i + 1];
				}
				stride[0] = stride[1] * inc[0];
				break;
			case CCV_NNC_MAX_DIM + 2:
				dim[0] = tensor->info.dim[CCV_NNC_MAX_DIM + 1];
				stride[0] = 1;
				dim[CCV_NNC_MAX_DIM + 1] = tensor->info.dim[CCV_NNC_MAX_DIM];
				stride[CCV_NNC_MAX_DIM + 1] = inc[CCV_NNC_MAX_DIM + 1];
				for (i = CCV_NNC_MAX_DIM; i > 0; i--)
				{
					dim[i] = tensor->info.dim[i - 1];
					stride[i] = stride[i + 1] * inc[i]; // inc[i] is actually the one before.
				}
				break;
			default:
				assert(0);
		}
	}
	if (CCV_NNC_MAX_DIM == 2)
	{
		CUDNN_ENFORCE(hipdnnSetTensor4dDescriptorEx(tensor_desc.descriptor, ccv_nnc_cudnn_datatype(tensor->info.datatype), dim[0], dim[1], dim[2], dim[3], stride[0], stride[1], stride[2], stride[3]));
	} else {
		CUDNN_ENFORCE(hipdnnSetTensorNdDescriptor(tensor_desc.descriptor, ccv_nnc_cudnn_datatype(tensor->info.datatype), CCV_NNC_MAX_DIM + 2, dim, stride));
	}
	return tensor_desc;
}

void ccv_nnc_cudnn_deinit_tensor_view_descriptor(const ccv_nnc_cudnn_tensor_view_descriptor_t tensor_desc)
{
	ccv_nnc_stream_context_return_tensor_descriptor(tensor_desc.stream_context, tensor_desc.descriptor);
}

ccv_nnc_cudnn_filter_descriptor_t ccv_nnc_cudnn_get_filter_descriptor(const ccv_nnc_stream_context_t* const stream_context, const ccv_nnc_tensor_t* const tensor)
{
	ccv_nnc_cudnn_filter_descriptor_t filter_desc = {
		stream_context,
		ccv_nnc_stream_context_get_filter_descriptor(stream_context),
		tensor->data,
	};
	assert(!CCV_IS_TENSOR_VIEW(tensor));
	const int nd = ccv_nnc_tensor_nd(tensor->info.dim);
	assert(nd == CCV_NNC_MAX_DIM + 2);
	int dim[CCV_NNC_MAX_DIM_ALLOC] = {};
	int i;
	if (tensor->info.format == CCV_TENSOR_FORMAT_NCHW)
	{
		for (i = 0; i < nd; i++)
			dim[i] = tensor->info.dim[i];
		if (nd == 4)
		{
			CUDNN_ENFORCE(hipdnnSetFilter4dDescriptor(filter_desc.descriptor, ccv_nnc_cudnn_datatype(tensor->info.datatype), HIPDNN_TENSOR_NCHW, dim[0], dim[1], dim[2], dim[3]));
		} else {
			CUDNN_ENFORCE(hipdnnSetFilterNdDescriptor(filter_desc.descriptor, ccv_nnc_cudnn_datatype(tensor->info.datatype), HIPDNN_TENSOR_NCHW, nd, dim));
		}
	} else if (tensor->info.format == CCV_TENSOR_FORMAT_NHWC) {
		dim[0] = tensor->info.dim[0];
		dim[1] = tensor->info.dim[nd - 1];
		for (i = 2; i < nd; i++)
			dim[i] = tensor->info.dim[i - 1];
		if (nd == 4)
		{
			CUDNN_ENFORCE(hipdnnSetFilter4dDescriptor(filter_desc.descriptor, ccv_nnc_cudnn_datatype(tensor->info.datatype), HIPDNN_TENSOR_NHWC, dim[0], dim[1], dim[2], dim[3]));
		} else {
			CUDNN_ENFORCE(hipdnnSetFilterNdDescriptor(filter_desc.descriptor, ccv_nnc_cudnn_datatype(tensor->info.datatype), HIPDNN_TENSOR_NHWC, nd, dim));
		}
	}
	return filter_desc;
}

void ccv_nnc_cudnn_deinit_filter_descriptor(const ccv_nnc_cudnn_filter_descriptor_t filter_desc)
{
	ccv_nnc_stream_context_return_filter_descriptor(filter_desc.stream_context, filter_desc.descriptor);
}

ccv_nnc_cudnn_convolution_descriptor_t ccv_nnc_cudnn_get_convolution_descriptor(const ccv_nnc_stream_context_t* const stream_context, const ccv_nnc_hint_t hint, const int datatype)
{
	ccv_nnc_cudnn_convolution_descriptor_t convolution_desc = {
		stream_context,
		ccv_nnc_stream_context_get_convolution_descriptor(stream_context),
	};
	int i;
	int p[CCV_NNC_MAX_DIM];
	for (i = 0; i < CCV_NNC_MAX_DIM; i++)
		p[i] = ccv_max(hint.border.begin[i], hint.border.end[i]);
	int v[CCV_NNC_MAX_DIM];
	for (i = 0; i < CCV_NNC_MAX_DIM; i++)
		v[i] = hint.stride.dim[i];
	if (CCV_NNC_MAX_DIM == 2)
	{
		CUDNN_ENFORCE(hipdnnSetConvolution2dDescriptor(convolution_desc.descriptor, p[0], p[1], v[0], v[1], 1, 1, HIPDNN_CROSS_CORRELATION, ccv_nnc_cudnn_datatype(datatype)));
	} else {
		int u[CCV_NNC_MAX_DIM];
		for (i = 0; i < CCV_NNC_MAX_DIM; i++)
			u[i] = 1;
		CUDNN_ENFORCE(hipdnnSetConvolutionNdDescriptor(convolution_desc.descriptor, CCV_NNC_MAX_DIM, p, v, u, HIPDNN_CROSS_CORRELATION, ccv_nnc_cudnn_datatype(datatype)));
	}
	CUDNN_ENFORCE(hipdnnSetConvolutionMathType(convolution_desc.descriptor, HIPDNN_TENSOR_OP_MATH));
	return convolution_desc;
}

void ccv_nnc_cudnn_deinit_convolution_descriptor(const ccv_nnc_cudnn_convolution_descriptor_t convolution_desc)
{
	ccv_nnc_stream_context_return_convolution_descriptor(convolution_desc.stream_context, convolution_desc.descriptor);
}
#endif

#ifdef HAVE_NCCL
static void _ccv_nnc_nccl_redo_comms(ncclComm_t* const comms, const int comm_count, const int device_count)
{
	int i;
	for (i = 0; i < comm_count; i++)
		NCCL_ENFORCE(ncclCommDestroy(comms[i]));
	int devs[device_count];
	for (i = 0; i < device_count; i++)
		devs[i] = i;
	NCCL_ENFORCE(ncclCommInitAll(comms, device_count, devs));
}

ncclComm_t ccv_nnc_nccl_get_comm(ccv_nnc_stream_context_t* const stream, const int device_count, const int device_id)
{
	assert(device_count > 0);
	if (stream)
	{
		ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream;
		if (!stream_compat->super.resource_container)
			stream_compat->super.resource_container = stream_compat->super._inline_container;
		if (!stream_compat->super.resource_container[0])
			stream_compat->super.resource_container[0] = (ccv_nnc_stream_resource_container_t*)cccalloc(1, sizeof(ccv_nnc_stream_resource_container_compat_t));
		ccv_nnc_stream_resource_container_compat_t* const resource_container_compat = (ccv_nnc_stream_resource_container_compat_t*)stream_compat->super.resource_container[0];
		if (resource_container_compat->comms && resource_container_compat->comm_count == device_count)
			return resource_container_compat->comms[device_id];
		if (resource_container_compat->comms)
			resource_container_compat->comms = (ncclComm_t*)ccrealloc(resource_container_compat->comms, sizeof(ncclComm_t) * device_count);
		else
			resource_container_compat->comms = (ncclComm_t*)ccmalloc(sizeof(ncclComm_t) * device_count);
		_ccv_nnc_nccl_redo_comms(resource_container_compat->comms, resource_container_compat->comm_count, device_count);
		resource_container_compat->comm_count = device_count;
		return resource_container_compat->comms[device_id];
	} else {
		static ncclComm_t comms[CCV_TENSOR_GET_DEVICE_ID(CCV_COMPUTE_DEVICE_ANY)];
		static int comm_count = 0;
		if (comm_count != device_count)
		{
			_ccv_nnc_nccl_redo_comms(comms, comm_count, device_count);
			comm_count = device_count;
		}
		return comms[device_id];
	}
}

ncclDataType_t ccv_nnc_nccl_datatype(const int datatype)
{
	switch (datatype)
	{
		case CCV_8U:
			return ncclUint8;
		case CCV_32S:
			return ncclInt;
		case CCV_16F:
			return ncclHalf;
		case CCV_32F:
			return ncclFloat;
		case CCV_64F:
			return ncclDouble;
	}
	return ncclFloat;
}
#endif
